#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

#include "simulate.hh"

using namespace std;


/* Utility function, use to do error checking for CUDA calls
 *
 * Use this function like this:
 *     checkCudaCall(<cuda_call>);
 *
 * For example:
 *     checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));
 * 
 * Special case to check the result of the last kernel invocation:
 *     kernel<<<...>>>(...);
 *     checkCudaCall(hipGetLastError());
**/

static void check(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(EXIT_FAILURE);
    }
}

// kernel computing each data point
__global__ void waveKernel(const long i_max, double *old, double *curr, double *next) {

    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < i_max) // if data is unevenly distributed, skip non-existing data
    next[i] = 2*curr[i] - old[i] + 0.15 * (curr[i-1] - (2*curr[i] - curr[i+1]));

    __syncthreads();

}

__constant__ double c = 0.15;

double *simulate(const long i_max, const long t_max, const long block_size,
                 double *old_array, double *current_array, double *next_array) {

    // init memory size of vectors
    int memSize = i_max * sizeof(double);
    // init devices
    double *deviceOld = NULL;
    double *deviceCurr = NULL;
    double *deviceNext = NULL;
    // allocate mem for devices
    check( hipMalloc((void **) &deviceOld,  memSize) );
    check( hipMalloc((void **) &deviceCurr, memSize) );
    check( hipMalloc((void **) &deviceNext, memSize) );
    // copy host arrays to GPU devices
    check( hipMemcpy(deviceOld,  old_array,     memSize, hipMemcpyHostToDevice) );
    check( hipMemcpy(deviceCurr, current_array, memSize, hipMemcpyHostToDevice) );
    check( hipMemcpy(deviceNext, next_array,    memSize, hipMemcpyHostToDevice) );

    // account for uneven distribution of threads
    int mod = i_max % block_size;
    if (mod != 0) mod = 1;
    int grid_size = i_max/block_size + mod;

    // calculate wave function
    for (int t = 0; t < t_max; t++) {
        
        // calc wave function
        waveKernel<<<grid_size, block_size>>>(i_max, deviceOld, deviceCurr, deviceNext);
        // swap buffers
        

    }
    
    // retrieve result from device to CPU
    check( hipMemcpy(next_array, deviceNext, memSize, hipMemcpyDeviceToHost) );

    hipFree(deviceOld);
    hipFree(deviceCurr);
    hipFree(deviceNext);

    
    return current_array;
}

// hipMemcpyToSymbol() for initializing c constant
// what's __shared__ used for? _syncthreads()
// why use __constant__ if you can just plug it in
// nvcc -Xptxas="-v" to see how many data per thread fits in fast registers
// prevent race conditions with atomics: order of operation still undefined though, good for counting
// copying data costs time, divide in chunks and let copying and processing go simultaneously